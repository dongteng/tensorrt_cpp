#include "hip/hip_runtime.h"
#include<iostream>
#include<vector>
#include <fstream>
#include <cassert>
#include"NvInfer.h"



class Logger : public nvinfer1::ILogger {
    void log(Severity severity, const char *msg) noexcept override {
        //����������ļ�һЩ����Severity::kWARNING
        if (severity <= Severity::kINFO)
            std::cout << msg << std::endl;
    }
} logger;
//����ģ��
std::vector<unsigned char> loadEngineModel(const std::string &fileName)
{
    std::ifstream file(fileName, std::ios::binary);        // �Զ����Ʒ�ʽ��ȡ
    assert(file.is_open() && "load engine model failed!"); // ����

    file.seekg(0, std::ios::end); // ��λ���ļ�ĩβ
    size_t size = file.tellg();   // ��ȡ�ļ���С

    std::vector<unsigned char> data(size); // ����һ��vector����СΪsize
    file.seekg(0, std::ios::beg);          // ��λ���ļ���ͷ
    file.read((char *)data.data(), size);  // ��ȡ�ļ����ݵ�data��
    file.close();

    return data;
}

int main() {
    // =========== 1 ����һ����������ʱruntime ===========
    Logger logger;
    nvinfer1::IRuntime *runtime = nvinfer1::createInferRuntime(logger);

    // =========== 2 �����л�����engine ============
    // ��ȡ�ļ�
    auto engineModel = loadEngineModel("/home/guest/user/zhjm/cppprojects/model_repo/model_demo.engine");
    nvinfer1::ICudaEngine *engine = runtime->deserializeCudaEngine(engineModel.data(), engineModel.size(), nullptr);

    if (!engine) {
        std::cout << "deserialize engine failed!" << std::endl;
        return -1;
    }

    // =========== 3 ����һ��ִ�������� ============
    nvinfer1::IExecutionContext* context = engine->createExecutionContext();
    if (!context) {
        std::cerr << "context create failed " << std::endl;
        return -1;
    }

    // ����������������״
    const int batch_size = 1; // ����ʵ���������
    const int sequence_length = 64; // ����ʵ���������
    const int input_size = batch_size * sequence_length;


    //����������״
    for (int inputIndex = 0; inputIndex < engine->getNbBindings(); ++inputIndex)
    {
        if (engine->bindingIsInput(inputIndex)) {
            context->setBindingDimensions(inputIndex, nvinfer1::Dims2(batch_size, sequence_length));
        }
    }
    std::vector<int64_t> inputIds = {101, 791, 1921, 1921, 3698, 2582, 720, 3416, 102, 0, 0, 0,0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                                           0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,0, 0, 0, 0};
    std::vector<int64_t> tokenTypeIds = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                                               0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    std::vector<int64_t> attentionMask = {1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                                                 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    // =========== 5 ����͸�������������ݵ��豸 ============
    // ��������� device �ڴ�
    void* gpuInputIds, *gpuTokenTypeId, *gpuAttentionMask, *gpuOutput0,*gpuOutput1;
    hipMalloc(&gpuInputIds, batch_size * sequence_length * sizeof(int64_t));
    hipMalloc(&gpuTokenTypeId, batch_size * sequence_length  * sizeof(int64_t));
    hipMalloc(&gpuAttentionMask, batch_size * sequence_length  * sizeof(int64_t));
    hipMalloc(&gpuOutput0, batch_size * sequence_length * 1024* sizeof(float));
    hipMalloc(&gpuOutput1, batch_size * 1024* sizeof(float));

    // �����ݴ��������Ƶ��豸
    hipMemcpyAsync(gpuInputIds, inputIds.data(), batch_size * sequence_length * sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpyAsync(gpuTokenTypeId, tokenTypeIds.data(), batch_size * sequence_length * sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpyAsync(gpuAttentionMask, attentionMask.data(), batch_size * sequence_length * sizeof(int64_t), hipMemcpyHostToDevice);


    // ===========  6 ִ������ ============
    void* bindings[] = {gpuInputIds, gpuTokenTypeId, gpuAttentionMask, gpuOutput0, gpuOutput1};
    bool success = context->enqueueV2( bindings, hipStreamDefault, nullptr);
    if (!success) {
        std::cerr << "enqueueV2 failed!" << std::endl;
        return -1;
    }

    // =========== 7 ����������ݵ����� ============
    std::vector<float> output0(1024);
    std::vector<float> output1(1024);
//    hipMemcpy(output0.data(), gpuOutput0, input_size * sizeof(float), hipMemcpyDeviceToHost); ��Ϊ����Ҫ���ֵ ���Դ���Դ�����ν

    hipMemcpy(output1.data(), gpuOutput1, 1024 * sizeof(float), hipMemcpyDeviceToHost);


    int i = 0;
    int a = 0;
    for (float val : output1) {
        i = i + 1;
        std::cout << val << ", ";

        if (val != 0.0) {
            a = a + 1;
        }
    }
    std::cout << "\nthe total mount of output " << i << std::endl;
    std::cout << "\n the  mount not 0 " << a << std::endl;
    // ===========  8 �ͷ���Դ  ============
    hipFree(gpuInputIds);
    hipFree(gpuTokenTypeId);
    hipFree(gpuAttentionMask);
    hipFree(gpuOutput1);
    hipFree(gpuOutput0);
    // ����TensorRT����
    context->destroy();
    engine->destroy();
    runtime->destroy();
    return 0;
}
